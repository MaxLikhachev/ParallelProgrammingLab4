#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <omp.h>

using namespace std;

#define BLOCK_SIZE 64
#define N 32

const float eps = 0.000001;

hipError_t calculateWithCuda(float* matrix, unsigned int size);

__global__ void globalCalculateKernel(float* c, float* a, float* b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    c[i * j] = sin(a[i * j]) * sin(a[i * j]) + cos(b[i * j]) * cos(b[i * j]) * cos(b[i * j]);
}

__global__ void sumKernel(float* matrix, float* sum, float* result, float* temp)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    // if (i != j)
        // sum += matrix[i * j] * result[j];

    // temp[i] = (matrix[i * size + size] - sum) / matrix[i * size + i];
    // c[i * j] = sin(a[i * j]) * sin(a[i * j]) + cos(b[i * j]) * cos(b[i * j]) * cos(b[i * j]);
}

bool isDiverged(float* result, float* temp, unsigned int size)
{
    bool flag = true;
    for (int i = 0; i < size && flag; i++)
        flag = fabs(temp[i] - result[i]) < eps;
    return flag;
}

bool isDiagonalDominanceBroken(float *matrix, unsigned int size)
{
    bool flag = true;
    for (int i = 0; i < size && flag; i++)
    {
        float fabsSum = 0.0;
        for (int j = 0; j < size; j++)
            if (i != j)
                fabsSum += fabs(matrix[i * size + j]);
        flag = fabs(matrix[i * size + i]) <= fabsSum;
    }
    return flag;
}


void initRandom(int arraySize, float* a)
{
    for (int i = 0; i < arraySize; i++)
        for (int j = 0; j < arraySize; j++)
        {
            a[i * arraySize + j] = 0 + rand() % arraySize;
            if (i == j) a[i * arraySize + j] += arraySize * arraySize;
        }       
}

void initMatrixNull(int arraySize, float* a)
{
    for (int i = 0; i < arraySize; i++)
        for (int j = 0; j < arraySize; j++)
            a[i * arraySize + j] = 0;
}

void initNull(int arraySize, float* a)
{
    for (int i = 0; i < arraySize; i++)
        a[i] = 0;
}

void display(int arraySize, float* a)
{
    for (int i = 0; i < arraySize; i++)
    {
        for (int j = 0; j < arraySize; j++)
            cout << a[i * arraySize + j] << " ";
        cout << endl;
    }
}

float* parallelOpenMPCalculate(float* matrix, unsigned int size)
{
    float* result = new float[size];
    float* temp = new float[size];
    initNull(size, result);
    initNull(size, temp);

    int count = 0;
    for (bool flag = !isDiagonalDominanceBroken(matrix, size); flag; count++)
    {
#pragma omp parallel for
        for (int i = 0; i < size; i++)
        {
            float sum = 0.0;
            {
#pragma omp parallel for
                for (int j = 0; j < size; j++)
                    if (i != j)
                        sum += matrix[i * size + j] * result[j];
                temp[i] = (matrix[i * size + size] - sum) / matrix[i * size + i];
            }
        }
        flag = !isDiverged(result, temp, size);
        if (flag)
            for (int i = 0; i < size; i++)
                result[i] = temp[i];
    }
    cout << "Parallel OpenMP count: " << count;
    return result;
}

float* sequentialCalculate(float* matrix, unsigned int size)
{
    float* result = new float[size];
    float* temp = new float[size];
    initNull(size, result);
    initNull(size, temp);
    
    int count = 0;
    for (bool flag = !isDiagonalDominanceBroken(matrix, size); flag; count++)
    {
        for (int i = 0; i < size; i++)
        {
            float sum = 0.0;
            for (int j = 0; j < size; j++)
                if (i != j)
                    sum += matrix[i * size + j] * result[j];
            temp[i] = (matrix[i * size + size] - sum) / matrix[i * size + i];
        }
        flag = !isDiverged(result, temp, size);
        if (flag)
            for (int i = 0; i < size; i++)
                result[i] = temp[i];
    }
    cout << "Sequential calculate count: " << count;
    return result;
}

int main()
{
    srand(time(NULL));

    cout << "Enter array size: ";
    int arraySize = 0;
    cin >> arraySize;

    float* matrix = new float[arraySize * arraySize];

    initRandom(arraySize, matrix);
    // display(arraySize, matrix);

    hipEvent_t start, stop;
    float KernelTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    sequentialCalculate(matrix, arraySize);

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&KernelTime, start, stop);
    printf("\nSequential calculate time:  %0.2f ms \n", KernelTime);

    hipEventRecord(start, 0);

    parallelOpenMPCalculate(matrix, arraySize);

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&KernelTime, start, stop);
    printf("\nParallel OpenMP calculate time:  %0.2f ms \n", KernelTime);

    // Add matrixes in parallel.
    /*
    hipError_t cudaStatus = calculateWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        cout << "calculateWithCuda failed!\n";
        return 1;
    }
  
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        cout << "hipDeviceReset failed!\n";
        return 1;
    }
    */
    return 0;
}


// Helper function for using CUDA to add matrixes in parallel.
hipError_t calculateWithCuda(float* matrix, unsigned int size)
{
    float* dev_matrix;

    hipError_t cudaStatus;
    hipEvent_t start, stop;
    float KernelTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for matrix.
    hipEventRecord(start, 0);
    cudaStatus = hipMalloc((void**)&dev_matrix, (N * N) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&KernelTime, start, stop);
    printf("\nAllocating GPU buffers time:  %0.2f ms \n", KernelTime);

    // Copy input matrixes from host memory to GPU buffers.
    hipEventRecord(start, 0);

    cudaStatus = hipMemcpy(dev_matrix, matrix, (N * N) * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&KernelTime, start, stop);
    printf("\nCopying matrix: host -> GPU  time:  %0.2f ms \n", KernelTime);

    // Launch a kernel on the GPU with one thread for each element.
    int numBlocks = BLOCK_SIZE;
    dim3 threadsPerBlock(N, N);
    cout << "\nConfig settings: arraySize = " << size << ", numBlocks = " << numBlocks << ", threadsPerBlock(" << N << ", " << N << ")\n";

    // Global memory
    hipEventRecord(start, 0);
    // globalCalculateKernel << <numBlocks, threadsPerBlock >> > (dev_c, dev_a, dev_b);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&KernelTime, start, stop);
    // cout << "\nGlobal result: " << KernelTime <<  " milliseconds\n";
    printf("\nGlobal memory work's time:  %0.2f ms \n", KernelTime);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "globalCalculateKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching globalCalculateKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output matrix from GPU buffer to host memory.
    hipEventRecord(start, 0);
    cudaStatus = hipMemcpy(matrix, dev_matrix, (N * N) * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&KernelTime, start, stop);
    printf("\nCopying output matri: GPU -> host time:  %0.2f ms \n", KernelTime);

Error:
    hipFree(dev_matrix);

    return cudaStatus;
}